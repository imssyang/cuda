﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"


__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = threadIdx.x;
    c[i] = a[i] * b[i] / 31415926;
    printf("----%d", i);
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size) {
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    int i = 0;
    hipError_t cudaStatus = hipErrorUnknown;

    if (hipSuccess != hipSetDevice(0)) goto Error; // Choose which GPU to run on
    if (hipSuccess != hipMalloc(&dev_c, size * sizeof(int))) goto Error; // Allocate GPU buffers for three vectors (two input, one output)
    if (hipSuccess != hipMalloc(&dev_a, size * sizeof(int))) goto Error;
    if (hipSuccess != hipMalloc(&dev_b, size * sizeof(int))) goto Error;
    if (hipSuccess != hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice)) goto Error; // Copy input vectors from host memory to GPU buffers
    if (hipSuccess != hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice)) goto Error;
    for (i = 0; i <= size/1024; i++)
        addKernel<<<1, 10>>>(dev_c, dev_a, dev_b); // Launch a kernel on the GPU with one thread for each element
        if (hipSuccess != hipGetLastError()) { printf("addKernel failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }
        if (hipSuccess != hipDeviceSynchronize()) goto Error; // Waits for the kernel to finish, and returns any errors encountered during the launch
    if (hipSuccess != hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost)) goto Error; // Copy output vector from GPU buffer to host memory
    cudaStatus = hipSuccess;

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return cudaStatus;
}

void runGPU() {
    const int size = 5;
    int* a = (int*)malloc(size * sizeof(int));
    int* b = (int*)malloc(size * sizeof(int));
    int* c = (int*)malloc(size * sizeof(int));
    int i = 0;

    for (i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 10;
    }

    if (hipSuccess != addWithCuda(c, a, b, size)) goto Error; // Add vectors in parallel
    printf("%u\n", c[size - 1]);

    if (hipSuccess != hipDeviceReset()) goto Error; // Called for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.

Error:
    free(a);
    free(b);
    free(c);
}

int main() {
    clock_t begin = clock();
    runGPU();
    clock_t end = clock();
    double times = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("time: %lf\n", times);
    return 0;
}
